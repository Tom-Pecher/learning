
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <hip/hip_runtime.h>

#define N 10000000  // Vector size = 10 million
#define BLOCK_SIZE 256

// Example:
// A = [1, 2, 3, 4, 5]
// B = [6, 7, 8, 9, 10]
// C = A + B = [7, 9, 11, 13, 15]

// Regular CPU vector addition
void vector_add_cpu(float *a, float *b, float *c, int n) {

    // Loop over vector elements and add them:
    for (int i = 0; i < n; i++) {
        c[i] = a[i] + b[i];
    }
}

// CUDA kernel for vector addition
__global__ void vector_add_gpu(float *a, float *b, float *c, int n) {

    // Get the index of the current thread:
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    // If thread is within the bounds of the vector, add the values:
    if (i < n) {
        c[i] = a[i] + b[i];
    }
}

// Initialize vector with random values
void init_vector(float *vec, int n) {
    for (int i = 0; i < n; i++) {
        vec[i] = (float)rand() / RAND_MAX;
    }
}

// Function to measure execution time
double get_time() {
    struct timespec ts;
    clock_gettime(CLOCK_MONOTONIC, &ts);
    return ts.tv_sec + ts.tv_nsec * 1e-9;
}

int main() {
    // All CPU (host) variables are designated with 'h', and all GPU (device) variables are designated with 'd':
    float *h_a, *h_b, *h_c_cpu, *h_c_gpu;
    float *d_a, *d_b, *d_c;
    size_t size = N * sizeof(float);

    // Allocate host memory
    h_a = (float*)malloc(size);
    h_b = (float*)malloc(size);
    h_c_cpu = (float*)malloc(size);
    h_c_gpu = (float*)malloc(size);

    // Initialize vectors
    srand(time(NULL));
    init_vector(h_a, N);
    init_vector(h_b, N);

    // llocate device memory
    hipMalloc(&d_a, size);
    hipMalloc(&d_b, size);
    hipMalloc(&d_c, size);

    // Copy data to device
    hipMemcpy(d_a, h_a, size, hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b, size, hipMemcpyHostToDevice);

    // Define grid and block dimensions
    int num_blocks = (N + BLOCK_SIZE - 1) / BLOCK_SIZE;
    // N = 1024, BLOCK_SIZE = 256, num_blocks = 4
    // (N + BLOCK_SIZE - 1) / BLOCK_SIZE = ( (1025 + 256 - 1) / 256 ) = 1280 / 256 = 4 rounded 

    // Warm-up runs
    printf("Performing warm-up runs...\n");
    for (int i = 0; i < 3; i++) {
        vector_add_cpu(h_a, h_b, h_c_cpu, N);
        vector_add_gpu<<<num_blocks, BLOCK_SIZE>>>(d_a, d_b, d_c, N);
        hipDeviceSynchronize();
    }

    // Benchmark CPU implementation
    printf("Benchmarking CPU implementation...\n");
    double cpu_total_time = 0.0;
    for (int i = 0; i < 20; i++) {
        double start_time = get_time();
        vector_add_cpu(h_a, h_b, h_c_cpu, N);
        double end_time = get_time();
        cpu_total_time += end_time - start_time;
    }
    double cpu_avg_time = cpu_total_time / 20.0;

    // Benchmark GPU implementation
    printf("Benchmarking GPU implementation...\n");
    double gpu_total_time = 0.0;
    for (int i = 0; i < 20; i++) {
        double start_time = get_time();
        vector_add_gpu<<<num_blocks, BLOCK_SIZE>>>(d_a, d_b, d_c, N);
        hipDeviceSynchronize();
        double end_time = get_time();
        gpu_total_time += end_time - start_time;
    }
    double gpu_avg_time = gpu_total_time / 20.0;

    // Print results
    printf("CPU average time: %f milliseconds\n", cpu_avg_time*1000);
    printf("GPU average time: %f milliseconds\n", gpu_avg_time*1000);
    printf("Speedup: %fx\n", cpu_avg_time / gpu_avg_time);

    // Verify results (optional)
    hipMemcpy(h_c_gpu, d_c, size, hipMemcpyDeviceToHost);
    bool correct = true;
    for (int i = 0; i < N; i++) {
        if (fabs(h_c_cpu[i] - h_c_gpu[i]) > 1e-5) {
            correct = false;
            break;
        }
    }
    printf("Results are %s\n", correct ? "correct" : "incorrect");

    // Free memory
    free(h_a);
    free(h_b);
    free(h_c_cpu);
    free(h_c_gpu);
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    return 0;
}