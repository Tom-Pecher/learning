
// This implementation uses 3D blocks.
// The point is that there are way more operations in this implementation, meaning it will slow down.
// Hence, use the 3D framework only when necessary.

#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <hip/hip_runtime.h>
#include <math.h>
#include <iostream>

#define N 10000000  // Vector size = 10 million
#define BLOCK_SIZE_1D 1024
#define BLOCK_SIZE_3D_X 16
#define BLOCK_SIZE_3D_Y 8
#define BLOCK_SIZE_3D_Z 8
// 16 * 16 * 8 = 2048

// CPU vector addition
void vector_add_cpu(float *a, float *b, float *c, int n) {
    for (int i = 0; i < n; i++) {
        c[i] = a[i] + b[i];
    }
}

// CUDA kernel for 1D vector addition
__global__ void vector_add_gpu_1d(float *a, float *b, float *c, int n) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    // one add, one multiply, one store
    if (i < n) {
        c[i] = a[i] + b[i];
        // one add, one store
    }
}

// CUDA kernel for 3D vector addition
__global__ void vector_add_gpu_3d(float *a, float *b, float *c, int nx, int ny, int nz) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    int k = blockIdx.z * blockDim.z + threadIdx.z;
    // 3 adds, 3 multiplies, 3 stores
    
    if (i < nx && j < ny && k < nz) {
        int idx = i + j * nx + k * nx * ny;
        if (idx < nx * ny * nz) {
            c[idx] = a[idx] + b[idx];
        }
    }
}

// Initialize vector with random values
void init_vector(float *vec, int n) {
    for (int i = 0; i < n; i++) {
        vec[i] = (float)rand() / RAND_MAX;
    }
}

// Function to measure execution time
double get_time() {
    struct timespec ts;
    clock_gettime(CLOCK_MONOTONIC, &ts);
    return ts.tv_sec + ts.tv_nsec * 1e-9;
}
int main() {
    float *h_a, *h_b, *h_c_cpu, *h_c_gpu_1d, *h_c_gpu_3d;
    float *d_a, *d_b, *d_c_1d, *d_c_3d;
    size_t size = N * sizeof(float);

    // Allocate host memory
    h_a = (float*)malloc(size);
    h_b = (float*)malloc(size);
    h_c_cpu = (float*)malloc(size);
    h_c_gpu_1d = (float*)malloc(size);
    h_c_gpu_3d = (float*)malloc(size);

    // Initialize vectors
    srand(time(NULL));
    init_vector(h_a, N);
    init_vector(h_b, N);

    // Allocate device memory
    hipMalloc(&d_a, size);
    hipMalloc(&d_b, size);
    hipMalloc(&d_c_1d, size);
    hipMalloc(&d_c_3d, size);

    // Copy data to device
    hipMemcpy(d_a, h_a, size, hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b, size, hipMemcpyHostToDevice);

    // Define grid and block dimensions for 1D
    int num_blocks_1d = (N + BLOCK_SIZE_1D - 1) / BLOCK_SIZE_1D;

    // Define grid and block dimensions for 3D
    int nx = 100, ny = 100, nz = 1000; // N = 10000000 = 100 * 100 * 1000
    dim3 block_size_3d(BLOCK_SIZE_3D_X, BLOCK_SIZE_3D_Y, BLOCK_SIZE_3D_Z);
    dim3 num_blocks_3d(
        (nx + block_size_3d.x - 1) / block_size_3d.x,
        (ny + block_size_3d.y - 1) / block_size_3d.y,
        (nz + block_size_3d.z - 1) / block_size_3d.z
    );

    // Warm-up runs
    printf("Performing warm-up runs...\n");
    for (int i = 0; i < 3; i++) {
        vector_add_cpu(h_a, h_b, h_c_cpu, N);
        vector_add_gpu_1d<<<num_blocks_1d, BLOCK_SIZE_1D>>>(d_a, d_b, d_c_1d, N);
        vector_add_gpu_3d<<<num_blocks_3d, block_size_3d>>>(d_a, d_b, d_c_3d, nx, ny, nz);
        hipDeviceSynchronize();
    }

    // Benchmark CPU implementation
    printf("Benchmarking CPU implementation...\n");
    double cpu_total_time = 0.0;
    for (int i = 0; i < 5; i++) {
        double start_time = get_time();
        vector_add_cpu(h_a, h_b, h_c_cpu, N);
        double end_time = get_time();
        cpu_total_time += end_time - start_time;
    }
    double cpu_avg_time = cpu_total_time / 5.0;

    // Benchmark GPU 1D implementation
    printf("Benchmarking GPU 1D implementation...\n");
    double gpu_1d_total_time = 0.0;
    for (int i = 0; i < 100; i++) {
        hipMemset(d_c_1d, 0, size);  // Clear previous results
        double start_time = get_time();
        vector_add_gpu_1d<<<num_blocks_1d, BLOCK_SIZE_1D>>>(d_a, d_b, d_c_1d, N);
        hipDeviceSynchronize();
        double end_time = get_time();
        gpu_1d_total_time += end_time - start_time;
    }
    double gpu_1d_avg_time = gpu_1d_total_time / 100.0;

    // Verify 1D results immediately
    hipMemcpy(h_c_gpu_1d, d_c_1d, size, hipMemcpyDeviceToHost);
    bool correct_1d = true;
    for (int i = 0; i < N; i++) {
        if (fabs(h_c_cpu[i] - h_c_gpu_1d[i]) > 1e-4) {
            correct_1d = false;
            std::cout << i << " cpu: " << h_c_cpu[i] << " != " << h_c_gpu_1d[i] << std::endl;
            break;
        }
    }
    printf("1D Results are %s\n", correct_1d ? "correct" : "incorrect");

    // Benchmark GPU 3D implementation
    printf("Benchmarking GPU 3D implementation...\n");
    double gpu_3d_total_time = 0.0;
    for (int i = 0; i < 100; i++) {
        hipMemset(d_c_3d, 0, size);  // Clear previous results
        double start_time = get_time();
        vector_add_gpu_3d<<<num_blocks_3d, block_size_3d>>>(d_a, d_b, d_c_3d, nx, ny, nz);
        hipDeviceSynchronize();
        double end_time = get_time();
        gpu_3d_total_time += end_time - start_time;
    }
    double gpu_3d_avg_time = gpu_3d_total_time / 100.0;

    // Verify 3D results immediately
    hipMemcpy(h_c_gpu_3d, d_c_3d, size, hipMemcpyDeviceToHost);
    bool correct_3d = true;
    for (int i = 0; i < N; i++) {
        if (fabs(h_c_cpu[i] - h_c_gpu_3d[i]) > 1e-4) {
            correct_3d = false;
            std::cout << i << " cpu: " << h_c_cpu[i] << " != " << h_c_gpu_3d[i] << std::endl;
            break;
        }
    }
    printf("3D Results are %s\n", correct_3d ? "correct" : "incorrect");

    // Print results
    printf("CPU average time: %f milliseconds\n", cpu_avg_time * 1000);
    printf("GPU 1D average time: %f milliseconds\n", gpu_1d_avg_time * 1000);
    printf("GPU 3D average time: %f milliseconds\n", gpu_3d_avg_time * 1000);
    printf("Speedup (CPU vs GPU 1D): %fx\n", cpu_avg_time / gpu_1d_avg_time);
    printf("Speedup (CPU vs GPU 3D): %fx\n", cpu_avg_time / gpu_3d_avg_time);
    printf("Speedup (GPU 1D vs GPU 3D): %fx\n", gpu_1d_avg_time / gpu_3d_avg_time);

    // Free memory
    free(h_a);
    free(h_b);
    free(h_c_cpu);
    free(h_c_gpu_1d);
    free(h_c_gpu_3d);
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c_1d);
    hipFree(d_c_3d);

    return 0;
}