

#include <hip/hip_runtime.h>
#include <stdio.h>

// A kernel is a special function that runs on your GPU (Graphics Card) instead of your CPU. 
// Think of it like giving instructions to a large team of workers (GPU threads) who can all work at the same time. 
// You mark a kernel with __global__ keyword, and it can only return void. Example:
__global__ void whoami(void) {
    // The grid represents the entire set of threads launched for a single kernel invocation. 
    // Think of it as the overall execution space. It's a collection of thread blocks.

    // A block is a group of threads that can cooperate and share data quickly through shared memory.
    // Threads within a block can share memory, synchronize with each other and cooperate on tasks:
    int block_id =
        blockIdx.x +
        blockIdx.y * gridDim.x +
        blockIdx.z * gridDim.x * gridDim.y;

    int block_offset =
        block_id *
        blockDim.x * blockDim.y * blockDim.z;

    // Warps are a group of threads that are executed together within a block.
    // Each warp is inside of a block and parallelizes 32 threads.
    // Threads in a warp share memory in an L1 cache so they can communicate quickly.
    // Instructions are issued to warps that then tell the threads what to do (not directly sent to threads).

    // The thread is the smallest unit of execution in CUDA. Each thread executes the kernel code independently. 
    // Within a block, threads are identified by a unique thread ID. 
    // This ID allows you to access specific data or perform different operations based on the thread's position within the block:
    int thread_offset =
        threadIdx.x +  
        threadIdx.y * blockDim.x +
        threadIdx.z * blockDim.x * blockDim.y;

    int id = block_offset + thread_offset; // the global id

    printf("%04d | Block(%d %d %d) = %3d | Thread(%d %d %d) = %3d\n",
        id,
        blockIdx.x, blockIdx.y, blockIdx.z, block_id,
        threadIdx.x, threadIdx.y, threadIdx.z, thread_offset);
}

int main(int argc, char **argv) {
    const int b_x = 2, b_y = 3, b_z = 4;
    const int t_x = 4, t_y = 4, t_z = 4; // the max warp size is 32, so 
    // we will get 2 warp of 32 threads per block

    int blocks_per_grid = b_x * b_y * b_z;
    int threads_per_block = t_x * t_y * t_z;

    printf("%d blocks/grid\n", blocks_per_grid);
    printf("%d threads/block\n", threads_per_block);
    printf("%d total threads\n", blocks_per_grid * threads_per_block);

    dim3 blocksPerGrid(b_x, b_y, b_z); // 3d cube of shape 2*3*4 = 24
    dim3 threadsPerBlock(t_x, t_y, t_z); // 3d cube of shape 4*4*4 = 64

    whoami<<<blocksPerGrid, threadsPerBlock>>>();
    hipDeviceSynchronize();
}
