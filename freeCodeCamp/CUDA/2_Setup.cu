

#include <hip/hip_runtime.h>
#include <iostream>

__global__ void hello_kernel() {
    printf("Hello, World! from GPU thread %d\n", threadIdx.x);
}

int main() {
    hello_kernel<<<1, 10>>>();
    hipDeviceSynchronize();
    
    return 0;
}
